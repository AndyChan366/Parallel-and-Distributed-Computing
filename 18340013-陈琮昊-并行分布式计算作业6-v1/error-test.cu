#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include "error_checks.h" // Macros CUDA_CHECK and CHECK_ERROR_MSG


__global__ void vector_add(double *C, const double *A, const double *B, int N)
{
    // Add the kernel code
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Do not try to access past the allocated memory
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}


int main(void)
{
    const int N = 20;
    const int ThreadsInBlock = 128;
    double *dA, *dB, *dC;
    double hA[N], hB[N], hC[N];
  
    for(int i = 0; i < N; ++i) {
        hA[i] = (double) i;
        hB[i] = (double) i * i;
    }

    /* 
       Add memory allocations and copies. Wrap your runtime function
       calls with CUDA_CHECK( ) macro
    */
    CUDA_CHECK( hipMalloc((void**)&dA, sizeof(double)*N) );
    CUDA_CHECK( hipMalloc((void**)&dB, sizeof(double)*N) );
    CUDA_CHECK( hipMalloc((void**)&dC, sizeof(double)*N) );
    CUDA_CHECK(hipMemcpy(dA, hA, sizeof(double) * N, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dB, hB, sizeof(double) * N, hipMemcpyHostToDevice));
    // #error Add the remaining memory allocations and copies

    // Note the maximum size of threads in a block
    dim3 grid, threads;

    //// Add the kernel call here
    vector_add<<<1, 32>>> (dC, dA, dB, N);
    // #error Add the CUDA kernel call


    // Here we add an explicit synchronization so that we catch errors
    // as early as possible. Don't do this in production code!
    hipDeviceSynchronize();
    CHECK_ERROR_MSG("vector_add kernel");

    //// Copy back the results and free the device memory
    CUDA_CHECK( hipMemcpy(hC, dC, sizeof(double) * N, hipMemcpyDeviceToHost));
    CUDA_CHECK( hipFree(dA));
    CUDA_CHECK( hipFree(dB));
    CUDA_CHECK( hipFree(dC));
    // #error Copy back the results and free the allocated memory

    for (int i = 0; i < N; i++)
        printf("%5.1f\n", hC[i]);

    return 0;
}